#include "hip/hip_runtime.h"
#include <Custom.h>

namespace Customlayer
{
template <unsigned nthdsPerCTA>
__launch_bounds__(nthdsPerCTA) __global__
    void pReLUKernel(const int n, const float negativeSlope, const float* input, float* output)
{
    for (int i = blockIdx.x * nthdsPerCTA + threadIdx.x; i < n; i += gridDim.x * nthdsPerCTA)
    {
        output[i] = input[i] > 0 ? input[i] : input[i] * negativeSlope;
    }
}

int lReLUGPU(hipStream_t stream, const int n, const float negativeSlope, const void* input, void* output)
{
    const int BS = 512;
    const int GS = (n + BS - 1) / BS;
    pReLUKernel<BS><<<GS, BS, 0, stream>>>(n, negativeSlope,
                                           (const float*) input,
                                           (float*) output);
    return 0;
}

int CustomInference(
    hipStream_t stream, const int n, const float negativeSlope, const void* input, void* output)
{
    return lReLUGPU(stream, n, negativeSlope, (const float*) input, (float*) output);
}
}
